#include "hip/hip_runtime.h"
/*
* TP 3 - Réduction CUDA
* --------------------------
* Mémoire paratagée, synchronisation, optimisation
*
* File: student.cu
* Author: Maxime MARIA
*/

#include "student.hpp"

namespace IMAC
{
	// ==================================================== EX 1
    __global__
    void maxReduce_ex1(const uint *const dev_array, const uint size, uint *const dev_partialMax)
	{	
		extern __shared__ uint values[];
		int idX = blockIdx.x * blockDim.x + threadIdx.x; int index = threadIdx.x;
		
		if (idX >= size) {
			return;
		}

		values[index] = dev_array[idX];
		__syncthreads();
		

		for(int i = 1; i < blockDim.x; i *= 2) {
			if (index + i < blockDim.x) {
				values[index] = umax(values[index], values[index + i]);
				
			}	
			__syncthreads();
		}
		
		if (threadIdx.x == 0)
			dev_partialMax[blockIdx.x] = values[index];
	}

	__global__
    void maxReduce_ex2(const uint *const dev_array, const uint size, uint *const dev_partialMax)
	{
		extern __shared__ uint values[];
		int idX = blockIdx.x * blockDim.x + threadIdx.x; int index = threadIdx.x;
		
		if (idX >= size) {
			return;
		}

		values[index] = dev_array[idX];
		__syncthreads();
		

		for(int i = blockDim.x / 2; i > 0; i /= 2) {
			if (index + i < blockDim.x) {
				values[index] = umax(values[index], values[index + i]);
				
			}	
			__syncthreads();
		}
		
		if (threadIdx.x == 0)
			dev_partialMax[blockIdx.x] = values[index];
	}

	__global__
	void maxReduce_ex3(const uint *const dev_array, const uint size, uint *const dev_partialMax)
	{
		extern __shared__ uint values[];
		int globalIndex = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
		int localIndex = threadIdx.x;

		uint val1 = (globalIndex < size) ? dev_array[globalIndex] : 0;
		uint val2 = (globalIndex + 1 < size) ? dev_array[globalIndex + 1] : 0;

		values[localIndex] = umax(val1, val2);
		__syncthreads();

		for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
			if (localIndex < stride) {
				values[localIndex] = umax(values[localIndex], values[localIndex + stride]);
			}
			__syncthreads();
		}

		if (localIndex == 0) {
			dev_partialMax[blockIdx.x] = values[0];
		}
	}

	__global__
void maxReduce_ex4(const uint *const dev_array, const uint size, uint *const dev_partialMax)
{
    extern __shared__ uint values[];
    int globalIndex = 2 * (blockIdx.x * blockDim.x + threadIdx.x); 
    int localIndex = threadIdx.x;

    uint val1 = (globalIndex < size) ? dev_array[globalIndex] : 0;
    uint val2 = (globalIndex + 1 < size) ? dev_array[globalIndex + 1] : 0;

    values[localIndex] = umax(val1, val2);
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 32; stride /= 2) {
        if (localIndex < stride) {
            values[localIndex] = umax(values[localIndex], values[localIndex + stride]);
        }
        __syncthreads();
    }

    volatile uint* vvalues = values;

    if (localIndex < 32) {
        vvalues[localIndex] = umax(vvalues[localIndex], vvalues[localIndex + 32]);
        vvalues[localIndex] = umax(vvalues[localIndex], vvalues[localIndex + 16]);
        vvalues[localIndex] = umax(vvalues[localIndex], vvalues[localIndex + 8]);
        vvalues[localIndex] = umax(vvalues[localIndex], vvalues[localIndex + 4]);
        vvalues[localIndex] = umax(vvalues[localIndex], vvalues[localIndex + 2]);
        vvalues[localIndex] = umax(vvalues[localIndex], vvalues[localIndex + 1]);
    }

    // Le thread 0 écrit le maximum partiel dans la mémoire globale
    if (localIndex == 0) {
        dev_partialMax[blockIdx.x] = values[0];
    }
}

	void studentJob(const std::vector<uint> &array, const uint resCPU /* Just for comparison */, const uint nbIterations)
    {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, 0);
		std::cout << "Max threads per block: " << prop.maxThreadsPerBlock << std::endl;
		
		uint *dev_array = NULL;
        const size_t bytes = array.size() * sizeof(uint);

		// Allocate array on GPU
		HANDLE_ERROR( hipMalloc( (void**)&dev_array, bytes ) );
		// Copy data from host to device
		HANDLE_ERROR( hipMemcpy( dev_array, array.data(), bytes, hipMemcpyHostToDevice ) );

		std::cout << "Test with " << nbIterations << " iterations" << std::endl;

		std::cout << "========== Ex 1 " << std::endl;
		uint res1 = 0; // result
		// Launch reduction and get timing
		float2 timing1 = reduce<KERNEL_EX1>(nbIterations, dev_array, array.size(), res1);
		
        std::cout << " -> Done: ";
        printTiming(timing1);
		compare(res1, resCPU); // Compare results

		std::cout << "========== Ex 2 " << std::endl;
		uint res2 = 0; // result
		// Launch reduction and get timing
		float2 timing2 = reduce<KERNEL_EX2>(nbIterations, dev_array, array.size(), res2);
		
        std::cout << " -> Done: ";
        printTiming(timing2);
		compare(res2, resCPU);

		std::cout << "========== Ex 3 " << std::endl;
		uint res3 = 0; // result
		// Launch reduction and get timing
		float2 timing3 = reduce<KERNEL_EX3>(nbIterations, dev_array, array.size(), res3);
		
        std::cout << " -> Done: ";
        printTiming(timing3);
		compare(res3, resCPU);

		std::cout << "========== Ex 4 " << std::endl;
		uint res4 = 0; // result
		// Launch reduction and get timing
		float2 timing4 = reduce<KERNEL_EX4>(nbIterations, dev_array, array.size(), res4);
		
        std::cout << " -> Done: ";
        printTiming(timing4);
		compare(res4, resCPU);

		std::cout << "========== Ex 5 " << std::endl;
		uint res5 = 0; // result
		// Launch reduction and get timing
		float2 timing5 = reduce<KERNEL_EX5>(nbIterations, dev_array, array.size(), res5);
		
        std::cout << " -> Done: ";
        printTiming(timing5);
		compare(res5, resCPU);

		// Free array on GPU
		hipFree( dev_array );
    }

	void printTiming(const float2 timing)
	{
		std::cout << ( timing.x < 1.f ? 1e3f * timing.x : timing.x ) << " us on device and ";
		std::cout << ( timing.y < 1.f ? 1e3f * timing.y : timing.y ) << " us on host." << std::endl;
	}

    void compare(const uint resGPU, const uint resCPU)
	{
		if (resGPU == resCPU)
		{
			std::cout << "Well done ! " << resGPU << " == " << resCPU << " !!!" << std::endl;
		}
		else
		{
			std::cout << "You failed ! " << resGPU << " != " << resCPU << " !!!" << std::endl;
		}
	}
}
