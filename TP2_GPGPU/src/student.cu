#include "hip/hip_runtime.h"
/*
* TP 2 - Convolution d'images
* --------------------------
* Mémoire constante et textures
*
* File: student.cu
* Author: Maxime MARIA
*/

#include "student.hpp"
#include "chronoGPU.hpp"

namespace IMAC
{
	__constant__
	float constant_mat[225]; // 15 * 15 = 225 max matrix size

	__device__
	float clampf_dev(const float val, const float min , const float max) 
	{
		return fminf(max, fmaxf(min, val));
	}

	__global__
	void conv_naive(const uchar4 *const input, const uint width, const uint height, const float *const mat, const uint matSize, uchar4 *const output) {
		for(uint y = blockIdx.y * blockDim.y + threadIdx.y; y < height; y += gridDim.y * blockDim.y) {
			for(uint x = blockIdx.x * blockDim.x + threadIdx.x; x < width; x += gridDim.x * blockDim.x) {
				float3 sum = make_float3(0.f,0.f,0.f);
				
				// Apply convolution
				for ( uint j = 0; j < matSize; ++j ) 
				{
					for ( uint i = 0; i < matSize; ++i ) 
					{
						int dX = x + i - matSize / 2;
						int dY = y + j - matSize / 2;

						// Handle borders
						if ( dX < 0 ) 
							dX = 0;

						if ( dX >= width ) 
							dX = width - 1;

						if ( dY < 0 ) 
							dY = 0;

						if ( dY >= height ) 
							dY = height - 1;

						const int idMat		= j * matSize + i;
						const int idPixel	= dY * width + dX;
						sum.x += (float)input[idPixel].x * mat[idMat];
						sum.y += (float)input[idPixel].y * mat[idMat];
						sum.z += (float)input[idPixel].z * mat[idMat];
					}
				}
				const int idOut = y * width + x;
				output[idOut].x = (uchar)clampf_dev( sum.x, 0.f, 255.f );
				output[idOut].y = (uchar)clampf_dev( sum.y, 0.f, 255.f );
				output[idOut].z = (uchar)clampf_dev( sum.z, 0.f, 255.f );
				output[idOut].w = 255;
			}
		}
	}

	__global__
	void conv_const(const uchar4 *const input, const uint width, const uint height, const uint matSize, uchar4 *const output) {
		for(uint y = blockIdx.y * blockDim.y + threadIdx.y; y < height; y += gridDim.y * blockDim.y) {
			for(uint x = blockIdx.x * blockDim.x + threadIdx.x; x < width; x += gridDim.x * blockDim.x) {
				float3 sum = make_float3(0.f,0.f,0.f);
				
				// Apply convolution
				for ( uint j = 0; j < matSize; ++j ) 
				{
					for ( uint i = 0; i < matSize; ++i ) 
					{
						int dX = x + i - matSize / 2;
						int dY = y + j - matSize / 2;

						// Handle borders
						if ( dX < 0 ) 
							dX = 0;

						if ( dX >= width ) 
							dX = width - 1;

						if ( dY < 0 ) 
							dY = 0;

						if ( dY >= height ) 
							dY = height - 1;

						const int idMat		= j * matSize + i;
						const int idPixel	= dY * width + dX;
						sum.x += (float)input[idPixel].x * constant_mat[idMat];
						sum.y += (float)input[idPixel].y * constant_mat[idMat];
						sum.z += (float)input[idPixel].z * constant_mat[idMat];
					}
				}
				const int idOut = y * width + x;
				output[idOut].x = (uchar)clampf_dev( sum.x, 0.f, 255.f );
				output[idOut].y = (uchar)clampf_dev( sum.y, 0.f, 255.f );
				output[idOut].z = (uchar)clampf_dev( sum.z, 0.f, 255.f );
				output[idOut].w = 255;
			}
		}
	}
	
	__global__
	void conv_1dtex(hipTextureObject_t texObj, const uint width, const uint height, const uint matSize, uchar4 *const output) {
		for(uint y = blockIdx.y * blockDim.y + threadIdx.y; y < height; y += gridDim.y * blockDim.y) {
			for(uint x = blockIdx.x * blockDim.x + threadIdx.x; x < width; x += gridDim.x * blockDim.x) {
				float3 sum = make_float3(0.f,0.f,0.f);
				
				// Apply convolution
				for ( uint j = 0; j < matSize; ++j ) 
				{
					for ( uint i = 0; i < matSize; ++i ) 
					{
						int dX = x + i - matSize / 2;
						int dY = y + j - matSize / 2;

						// Handle borders
						if ( dX < 0 ) 
							dX = 0;

						if ( dX >= width ) 
							dX = width - 1;

						if ( dY < 0 ) 
							dY = 0;

						if ( dY >= height ) 
							dY = height - 1;

						// Compute normalized coordinates
						float texCoord = dY * width + dX;

						// Fetch from texture
						uchar4 pixel = tex1Dfetch<uchar4>(texObj, texCoord);
						
						const int idMat = j * matSize + i;
						sum.x += (float)pixel.x * constant_mat[idMat];
						sum.y += (float)pixel.y * constant_mat[idMat];
						sum.z += (float)pixel.z * constant_mat[idMat];

						/*const int idPixel	= dY * width + dX;
						sum.x += (float)input[idPixel].x * constant_mat[idMat];
						sum.y += (float)input[idPixel].y * constant_mat[idMat];
						sum.z += (float)input[idPixel].z * constant_mat[idMat];*/
					}
				}
				const int idOut = y * width + x;
				output[idOut].x = (uchar)clampf_dev( sum.x, 0.f, 255.f );
				output[idOut].y = (uchar)clampf_dev( sum.y, 0.f, 255.f );
				output[idOut].z = (uchar)clampf_dev( sum.z, 0.f, 255.f );
				output[idOut].w = 255;
			}
		}
	}

	__global__
	void conv_2dtex(hipTextureObject_t texObj, const uint width, const uint height, const uint matSize, uchar4 *const output) {
		for(uint y = blockIdx.y * blockDim.y + threadIdx.y; y < height; y += gridDim.y * blockDim.y) {
			for(uint x = blockIdx.x * blockDim.x + threadIdx.x; x < width; x += gridDim.x * blockDim.x) {
				float3 sum = make_float3(0.f,0.f,0.f);
				
				// Apply convolution
				for ( uint j = 0; j < matSize; ++j ) 
				{
					for ( uint i = 0; i < matSize; ++i ) 
					{
						int dX = x + i - matSize / 2;
						int dY = y + j - matSize / 2;

						// Handle borders
						if ( dX < 0 ) 
							dX = 0;

						if ( dX >= width ) 
							dX = width - 1;

						if ( dY < 0 ) 
							dY = 0;

						if ( dY >= height ) 
							dY = height - 1;

						// Compute normalized coordinates
						int texCoord = dY * width + dX;

						// Fetch from texture
						uchar4 pixel = tex2D<uchar4>(texObj, texCoord % width, texCoord / height);
						
						const int idMat = j * matSize + i;
						sum.x += (float)pixel.x * constant_mat[idMat];
						sum.y += (float)pixel.y * constant_mat[idMat];
						sum.z += (float)pixel.z * constant_mat[idMat];

						/*const int idPixel	= dY * width + dX;
						sum.x += (float)input[idPixel].x * constant_mat[idMat];
						sum.y += (float)input[idPixel].y * constant_mat[idMat];
						sum.z += (float)input[idPixel].z * constant_mat[idMat];*/
					}
				}
				const int idOut = y * width + x;
				output[idOut].x = (uchar)clampf_dev( sum.x, 0.f, 255.f );
				output[idOut].y = (uchar)clampf_dev( sum.y, 0.f, 255.f );
				output[idOut].z = (uchar)clampf_dev( sum.z, 0.f, 255.f );
				output[idOut].w = 255;
			}
		}
	}

// ================================================== For image comparison
	std::ostream &operator <<(std::ostream &os, const uchar4 &c)
	{
		os << "[" << uint(c.x) << "," << uint(c.y) << "," << uint(c.z) << "," << uint(c.w) << "]";  
    	return os; 
	}

	void compareImages(const std::vector<uchar4> &a, const std::vector<uchar4> &b)
	{
		bool error = false;
		if (a.size() != b.size())
		{
			std::cout << "Size is different !" << std::endl;
			error = true;
		}
		else
		{
			for (uint i = 0; i < a.size(); ++i)
			{
				// Floating precision can cause small difference between host and device
				if (	std::abs(a[i].x - b[i].x) > 2 || std::abs(a[i].y - b[i].y) > 2 
					|| std::abs(a[i].z - b[i].z) > 2 || std::abs(a[i].w - b[i].w) > 2)
				{
					std::cout << "Error at index " << i << ": a = " << a[i] << " - b = " << b[i] << " - " << std::abs(a[i].x - b[i].x) << std::endl;
					error = true;
					break;
				}
			}
		}
		if (error)
		{
			std::cout << " -> You failed, retry!" << std::endl;
		}
		else
		{
			std::cout << " -> Well done!" << std::endl;
		}
	}
	
	// ==================================================
    void studentJob_Naive(const std::vector<uchar4> &inputImg, // Input image
					const uint imgWidth, const uint imgHeight, // Image size
                    const std::vector<float> &matConv, // Convolution matrix (square)
					const uint matSize, // Matrix size (width or height)
					const std::vector<uchar4> &resultCPU, // Just for comparison
                    std::vector<uchar4> &output // Output image
					)
	{
		std::cout << "====================================================== Naive" << std::endl;

		ChronoGPU chrGPU;

		uchar4 *dev_input = NULL;
		uchar4 *dev_output = NULL;
		float *dev_mat = NULL;

		const size_t bytes = inputImg.size() * sizeof(uchar4);
		const size_t bytes_mat = matConv.size() * sizeof(float);

		std::cout 	<< "Allocating input (2 arrays) and matrix: " 
					<< ( ( 2 * bytes + bytes_mat ) >> 20 ) << " MB on Device" << std::endl;
		chrGPU.start();		
		HANDLE_ERROR( hipMalloc( (void**)&dev_input, bytes ) );
		HANDLE_ERROR( hipMalloc( (void**)&dev_output, bytes ) );	
		HANDLE_ERROR( hipMalloc( (void**)&dev_mat, bytes_mat ) );	
		chrGPU.stop();
		std::cout 	<< "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		// Copy data from host to device (input array) 
		std::cout << "Copy input from host to device" << std::endl;
		chrGPU.start();		
		HANDLE_ERROR( hipMemcpy( dev_input, inputImg.data(), bytes, hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy( dev_mat, matConv.data(), bytes_mat, hipMemcpyHostToDevice ) );
		chrGPU.stop();
		std::cout 	<< "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		const dim3 threads(32, 32); // 32 * 32 = 1024
		const dim3 blocks( ( imgWidth + threads.x - 1 ) / threads.x, ( imgHeight + threads.y - 1 ) / threads.y );

		std::cout << "-> Size of blocks x : " << blocks.x << " || Size of blocks y : " << blocks.y << std::endl;
		
		// Launch kernel
		std::cout << "Naive Convolution on GPU (" 	<< blocks.x << "x" << blocks.y << " blocks - " 
												<< threads.x << "x" << threads.y << " threads)" << std::endl;
		chrGPU.start();
		conv_naive<<<blocks, threads>>>(dev_input, imgWidth, imgHeight, dev_mat, matSize, dev_output);
		chrGPU.stop();
		std::cout 	<< "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		// Copy data from device to host (output array)   
		std::cout << "Copy output from device to host" << std::endl;
		chrGPU.start();
		HANDLE_ERROR( hipMemcpy( output.data(), dev_output, bytes, hipMemcpyDeviceToHost ) ); 
		chrGPU.stop();
		std::cout 	<< "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl;

		// Free arrays on device
		std::cout << "Free memory on GPU" << std::endl;
		hipFree(dev_input);
		hipFree(dev_output);
		hipFree(dev_mat);

		std::cout << "Comparison of final results" << std::endl;
		compareImages(resultCPU, output);

		std::cout << "============================================================" << std::endl << std::endl;
	}
	
    void studentJob_Constant(const std::vector<uchar4> &inputImg, // Input image
					const uint imgWidth, const uint imgHeight, // Image size
                    const std::vector<float> &matConv, // Convolution matrix (square)
					const uint matSize, // Matrix size (width or height)
					const std::vector<uchar4> &resultCPU, // Just for comparison
                    std::vector<uchar4> &output // Output image
					)
	{
		std::cout << "====================================================== Constant" << std::endl;
		
		ChronoGPU chrGPU;

		uchar4 *dev_input = NULL;
		uchar4 *dev_output = NULL;

		const size_t bytes = inputImg.size() * sizeof(uchar4);
		const size_t bytes_mat = matConv.size() * sizeof(float);

		std::cout 	<< "Allocating input (2 arrays) and matrix: " 
					<< ( ( 2 * bytes + bytes_mat ) >> 20 ) << " MB on Device" << std::endl;
		chrGPU.start();		
		HANDLE_ERROR( hipMalloc( (void**)&dev_input, bytes ) );
		HANDLE_ERROR( hipMalloc( (void**)&dev_output, bytes ) );	
		chrGPU.stop();
		std::cout 	<< "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		// Copy data from host to device (input array) 
		std::cout << "Copy input from host to device" << std::endl;
		chrGPU.start();		
		HANDLE_ERROR( hipMemcpy( dev_input, inputImg.data(), bytes, hipMemcpyHostToDevice ) );
		// Copy the host array to device constant memory
    	HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( constant_mat), matConv.data(), bytes_mat ) );
		
		chrGPU.stop();
		std::cout 	<< "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		const dim3 threads(32, 32); // 32 * 32 = 1024
		const dim3 blocks( ( imgWidth + threads.x - 1 ) / threads.x, ( imgHeight + threads.y - 1 ) / threads.y );

		std::cout << "-> Size of blocks x : " << blocks.x << " || Size of blocks y : " << blocks.y << std::endl;
		
		// Launch kernel
		std::cout << "Constant Convolution on GPU (" 	<< blocks.x << "x" << blocks.y << " blocks - " 
												<< threads.x << "x" << threads.y << " threads)" << std::endl;
		chrGPU.start();
		conv_const<<<blocks, threads>>>(dev_input, imgWidth, imgHeight, matSize, dev_output);
		chrGPU.stop();
		std::cout 	<< "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		// Copy data from device to host (output array)   
		std::cout << "Copy output from device to host" << std::endl;
		chrGPU.start();
		HANDLE_ERROR( hipMemcpy( output.data(), dev_output, bytes, hipMemcpyDeviceToHost ) ); 		
		chrGPU.stop();
		std::cout 	<< "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl;

		// Free arrays on device
		std::cout << "Free memory on GPU" << std::endl;
		hipFree(dev_input);
		hipFree(dev_output);

		std::cout << "Comparison of final results" << std::endl;
		compareImages(resultCPU, output);

		std::cout << "============================================================" << std::endl << std::endl;
	}
	
    void studentJob_Tex1D(const std::vector<uchar4> &inputImg, // Input image
					const uint imgWidth, const uint imgHeight, // Image size
                    const std::vector<float> &matConv, // Convolution matrix (square)
					const uint matSize, // Matrix size (width or height)
					const std::vector<uchar4> &resultCPU, // Just for comparison
                    std::vector<uchar4> &output // Output image
					)
	{
		std::cout << "====================================================== Texture 1D" << std::endl;
		ChronoGPU chrGPU;

		// hipTextureObject_t *pTexObject, const hipResourceDesc *pResDesc, const hipTextureDesc *pTexDesc, const hipResourceViewDesc *pResViewDesc
		//hipCreateTextureObject() 

		uchar4 *dev_input = NULL;
		uchar4 *dev_output = NULL;

		const size_t bytes = inputImg.size() * sizeof(uchar4);
		const size_t bytes_mat = matConv.size() * sizeof(float);

		std::cout 	<< "Allocating input (2 arrays) and matrix: " 
					<< ( ( 2 * bytes + bytes_mat ) >> 20 ) << " MB on Device" << std::endl;
		chrGPU.start();		
		HANDLE_ERROR( hipMalloc( (void**)&dev_input, bytes ) );
		HANDLE_ERROR( hipMalloc( (void**)&dev_output, bytes ) );	
		chrGPU.stop();
		std::cout 	<< "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		// Copy data from host to device (input array) 
		std::cout << "Copy input from host to device" << std::endl;
		chrGPU.start();		
		HANDLE_ERROR( hipMemcpy( dev_input, inputImg.data(), bytes, hipMemcpyHostToDevice ) );
		// Copy the host array to device constant memory
    	HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( constant_mat), matConv.data(), bytes_mat ) );
		chrGPU.stop();
		std::cout 	<< "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		// Create CUDA array for texture binding
		hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar4>();

		// Init texture ressources
		struct hipResourceDesc resDesc {};
		resDesc.resType = hipResourceTypeLinear;
		resDesc.res.linear.devPtr = dev_input;
		resDesc.res.linear.sizeInBytes = bytes;
		resDesc.res.linear.desc = channelDesc;

		// Init texture descriptor
		struct hipTextureDesc texDesc {};
		texDesc.addressMode[0] = hipAddressModeClamp;
		texDesc.addressMode[1] = hipAddressModeClamp;
		texDesc.filterMode = hipFilterModePoint;
		texDesc.readMode = hipReadModeElementType;
		//texDesc.normalizedCoords = 0;

		// Init texture object
		hipTextureObject_t texObj = 0;
		hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);

		const dim3 threads(32, 32); // 32 * 32 = 1024
		const dim3 blocks( ( imgWidth + threads.x - 1 ) / threads.x, ( imgHeight + threads.y - 1 ) / threads.y );

		std::cout << "-> Size of blocks x : " << blocks.x << " || Size of blocks y : " << blocks.y << std::endl;
		
		// Launch kernel
		std::cout << "Tex1D Convolution on GPU (" 	<< blocks.x << "x" << blocks.y << " blocks - " 
												<< threads.x << "x" << threads.y << " threads)" << std::endl;
		chrGPU.start();
		conv_1dtex<<<blocks, threads>>>(texObj, imgWidth, imgHeight, matSize, dev_output);
		chrGPU.stop();
		std::cout 	<< "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		// Copy data from device to host (output array)   
		std::cout << "Copy output from device to host" << std::endl;
		chrGPU.start();
		HANDLE_ERROR( hipMemcpy( output.data(), dev_output, bytes, hipMemcpyDeviceToHost ) ); 		
		chrGPU.stop();
		std::cout 	<< "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl;

		// Free arrays on device
		std::cout << "Free memory on GPU" << std::endl;
		hipDestroyTextureObject(texObj);
		hipFree(dev_input);
		hipFree(dev_output);

		std::cout << "Comparison of final results" << std::endl;
		compareImages(resultCPU, output);

		std::cout << "============================================================" << std::endl << std::endl;
	}
	
    void studentJob_Tex2D(const std::vector<uchar4> &inputImg, // Input image
					const uint imgWidth, const uint imgHeight, // Image size
                    const std::vector<float> &matConv, // Convolution matrix (square)
					const uint matSize, // Matrix size (width or height)
					const std::vector<uchar4> &resultCPU, // Just for comparison
                    std::vector<uchar4> &output // Output image
					)
	{
		std::cout << "====================================================== Texture 2D" << std::endl;
		
		std::cout << "====================================================== Texture 1D" << std::endl;
		ChronoGPU chrGPU;

		// hipTextureObject_t *pTexObject, const hipResourceDesc *pResDesc, const hipTextureDesc *pTexDesc, const hipResourceViewDesc *pResViewDesc
		//hipCreateTextureObject() 

		uchar4 *dev_input = NULL;
		uchar4 *dev_output = NULL;

		const size_t bytes = inputImg.size() * sizeof(uchar4);
		const size_t bytes_mat = matConv.size() * sizeof(float);

		std::cout 	<< "Allocating input (2 arrays) and matrix: " 
					<< ( ( 2 * bytes + bytes_mat ) >> 20 ) << " MB on Device" << std::endl;
		chrGPU.start();		
		HANDLE_ERROR( hipMalloc( (void**)&dev_input, bytes ) );
		HANDLE_ERROR( hipMalloc( (void**)&dev_output, bytes ) );	
		chrGPU.stop();
		std::cout 	<< "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		// Copy data from host to device (input array) 
		std::cout << "Copy input from host to device" << std::endl;
		chrGPU.start();		
		HANDLE_ERROR( hipMemcpy( dev_input, inputImg.data(), bytes, hipMemcpyHostToDevice ) );
		// Copy the host array to device constant memory
    	HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( constant_mat), matConv.data(), bytes_mat ) );
		chrGPU.stop();
		std::cout 	<< "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		// Create CUDA array for texture binding
		hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar4>();
		uchar4 *dev_array = NULL;
		size_t pitch;
		//hipMallocArray(&dev_array, &channelDesc, bytes);
		hipMallocPitch(&dev_array, &pitch, imgWidth * sizeof(uchar4), imgHeight);

		// Copy device memory to CUDA array
		//hipMemcpyToArray(dev_array, 0, 0, dev_input, bytes, hipMemcpyDeviceToDevice);
		hipMemcpy2D(dev_array, pitch, inputImg.data(), imgWidth * sizeof(uchar4), imgWidth * sizeof(uchar4), imgHeight, hipMemcpyHostToDevice);

		// Free the device buffer because data is now in CUDA Array
		hipFree(dev_input);

		// Init texture ressources
		struct hipResourceDesc resDesc {};
		resDesc.resType = hipResourceTypePitch2D;
		resDesc.res.pitch2D.desc = hipCreateChannelDesc<uchar4>();
		resDesc.res.pitch2D.devPtr = dev_array;
		resDesc.res.pitch2D.height = imgHeight;
		resDesc.res.pitch2D.width = imgWidth;
		resDesc.res.pitch2D.pitchInBytes = pitch;

		// Init texture descriptor
		struct hipTextureDesc texDesc {};
		texDesc.addressMode[0] = hipAddressModeClamp;
		texDesc.filterMode = hipFilterModePoint;
		texDesc.readMode = hipReadModeElementType;

		// Init texture object
		hipTextureObject_t texObj = 0;
		hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);

		const dim3 threads(32, 32); // 32 * 32 = 1024
		const dim3 blocks( ( imgWidth + threads.x - 1 ) / threads.x, ( imgHeight + threads.y - 1 ) / threads.y );

		std::cout << "-> Size of blocks x : " << blocks.x << " || Size of blocks y : " << blocks.y << std::endl;
		
		// Launch kernel
		std::cout << "Tex2D Convolution on GPU (" 	<< blocks.x << "x" << blocks.y << " blocks - " 
												<< threads.x << "x" << threads.y << " threads)" << std::endl;
		chrGPU.start();
		conv_2dtex<<<blocks, threads>>>(texObj, imgWidth, imgHeight, matSize, dev_output);
		chrGPU.stop();
		std::cout 	<< "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		// Copy data from device to host (output array)   
		std::cout << "Copy output from device to host" << std::endl;
		chrGPU.start();
		HANDLE_ERROR( hipMemcpy( output.data(), dev_output, bytes, hipMemcpyDeviceToHost ) ); 		
		chrGPU.stop();
		std::cout 	<< "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl;

		// Free arrays on device
		std::cout << "Free memory on GPU" << std::endl;
		hipDestroyTextureObject(texObj);
		hipFree(dev_output);

		std::cout << "Comparison of final results" << std::endl;
		compareImages(resultCPU, output);

		std::cout << "============================================================" << std::endl << std::endl;
	}
	
    void studentJob(const std::vector<uchar4> &inputImg, // Input image
					const uint imgWidth, const uint imgHeight, // Image size
                    const std::vector<float> &matConv, // Convolution matrix (square)
					const uint matSize, // Matrix size (width or height)
					const std::vector<uchar4> &resultCPU, // Just for comparison
                    std::vector<uchar4> &output // Output image
					)
	{
		studentJob_Naive(inputImg,imgWidth,imgHeight,matConv,matSize,resultCPU,output);
		studentJob_Constant(inputImg,imgWidth,imgHeight,matConv,matSize,resultCPU,output);
		studentJob_Tex1D(inputImg,imgWidth,imgHeight,matConv,matSize,resultCPU,output);
		studentJob_Tex2D(inputImg,imgWidth,imgHeight,matConv,matSize,resultCPU,output);		
	}
}
